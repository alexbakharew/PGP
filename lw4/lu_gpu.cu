#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <string.h>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#define THREADS_PER_BLOCK 20
#define BLOCKS_PER_GRID 20

#define CSC(call)  					\
do {								\
	hipError_t res = call;			\
	if (res != hipSuccess) {		\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);					\
	}								\
} while(0)

__host__ void cpu_swap(double* lhs, double* rhs)
{
	double tmp = *lhs;
	*lhs = *rhs;
	*rhs = tmp;
}

__host__ void cpu_transpose(double* matrix, int size)
{
	for (int i = 0; i < size; ++i)
	{
		for (int j = i + 1; j < size; ++j)
		{
			cpu_swap(&matrix[i * size + j], &matrix[j * size + i]);
		}
	}
}   

__host__ void gpu_print_matrix(double* matrix, int size)
{
	for (int i = 0; i < size; ++i)
	{
		for (int j = 0; j < size; ++j)
		{
			printf("%.1f ", matrix[i * size + j]);
		}
		printf("\n");
	}
}

__global__ void gpu_transpose(double* matrix, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offsetx = gridDim.x * blockDim.x;
    double temp;
    int curr_row;
    int curr_col;
    while(idx < size * size)
    {
        curr_row = idx / size;
        curr_col = idx % size;
        if(curr_col > curr_row)
        {
            temp = matrix[curr_row * size + curr_col];
            matrix[curr_row * size + curr_col] = matrix[curr_col * size + curr_row];
            matrix[curr_col * size + curr_row] = temp;
        }
	    idx += offsetx;
        
	}
}

__global__ void gpu_swap(double* matrix, int size, int row_from, int row_to)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offsetx = gridDim.x * blockDim.x;
	double tmp;
	for (int i = idx; i < size; i += offsetx)
	{
		tmp = matrix[(i * size) + row_from];
		matrix[(i * size) + row_from] = matrix[(i * size) + row_to];
		matrix[(i * size) + row_to] = tmp;
	}
}

double* multiplication(double* lhs, double* rhs, int n)
{
    double* res = (double*) malloc(sizeof(double) * n * n);

    res = (double*) calloc(n * n, sizeof(double));        


    for(int i = 0; i < n; ++i)
    {
        for(int j = 0; j < n; ++j)
        {        
            for(int t = 0; t < n; ++t)
            {
                res[j * n + i] += lhs[j * n + t] * rhs[t * n + i];
            }
        }
    }
    return res;
}

__global__ void gpu_compute_L(double* matrix, double* L, int size, int curr_row)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;//shift from main diagonal
	int offsetx = gridDim.x * blockDim.x;

	for(; idx < size; idx += offsetx)
	{
		if(idx < curr_row)
			continue;

		if(idx == curr_row )
		{
			L[curr_row * size + curr_row] = 1.0;
		}

		else if(matrix[curr_row * size + curr_row] != 0)
		{
			L[curr_row * size + idx] = matrix[curr_row * size + idx] / matrix[curr_row * size + curr_row];
		}
	}
}

__global__ void modify_matrix(double* matrix, double* L, int size, int max_col)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offsetx = gridDim.x * blockDim.x;
	int curr_row;
	int curr_col;
	for(; idx < size * size; idx += offsetx)
	{
		curr_row = idx / size;
		curr_col = idx % size;
		if(curr_col == max_col)
			continue;
		else
		{
			//matrix[idx] -= L[max_col * size + curr_col] * matrix[curr_row * size + max_col];
			matrix[curr_row * size + curr_col] -= L[max_col * size + curr_col] * matrix[curr_row * size + max_col];
			
		}
	}
}
int main()
{
	int n;
	scanf("%d", &n);
	double* matrix = (double*)malloc(sizeof(double) * n * n);
	for (int i = 0; i < n * n; ++i)
	{
		scanf("%lf", &matrix[i]);
	}
	double* matrix_dev;
	CSC(hipMalloc(&matrix_dev, sizeof(double) * n * n));
	CSC(hipMemcpy(matrix_dev, matrix, sizeof(double) * n * n, hipMemcpyHostToDevice));

	// printf("matrix------------\n");
	// gpu_print_matrix(matrix, n);
	// printf("\n");

	gpu_transpose << <32, 32 >> > (matrix_dev, n);

	hipMemcpy(matrix, matrix_dev, sizeof(double) * n * n, hipMemcpyDeviceToHost);
	printf("transposed matrix------------\n");
	gpu_print_matrix(matrix, n);

	double* L = (double*) calloc(n * n, sizeof(double));
	double* L_dev;
	CSC(hipMalloc(&L_dev, sizeof(double) * n * n));
	CSC(hipMemcpy(L_dev, L, sizeof(double) * n * n, hipMemcpyHostToDevice));

	int* P = (int*) malloc(sizeof(int) * n);
	
	// printf("jopa");

	// thrust::device_ptr<double> max_elem;
	int pos_of_max;
	int sign = 1;
	// getchar();

	for (int row = 0; row < n; ++row)
	{
		thrust::device_ptr<double> p_matrix = thrust::device_pointer_cast(matrix_dev);
		thrust::device_ptr<double> max_elem = thrust::max_element(p_matrix + (row * n) + row, p_matrix + ((row + 1) * n));
		pos_of_max = (int)(max_elem - p_matrix) % n;
		//printf("%d\n", pos_of_max);
		//P[row] = pos_of_max;

		if(row != pos_of_max)
		{
			sign *= -1;
			gpu_swap<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>> (matrix_dev, n, row, pos_of_max);
			CSC(hipMemcpy(matrix, matrix_dev, sizeof(double) * n * n, hipMemcpyDeviceToHost));
			printf("U after swap-----\n");
			gpu_print_matrix(matrix, n);
			
		}

		gpu_compute_L << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> > (matrix_dev, L_dev, n, row);
		CSC(hipMemcpy(L, L_dev, sizeof(double) * n * n, hipMemcpyDeviceToHost));
		printf("L-----\n");
		gpu_print_matrix(L, n);
		getchar();

		modify_matrix <<< BLOCKS_PER_GRID, THREADS_PER_BLOCK >>> (matrix_dev, L_dev, n, row);
		CSC(hipMemcpy(matrix, matrix_dev, sizeof(double) * n * n, hipMemcpyDeviceToHost));
		printf("U-----\n");
		gpu_print_matrix(matrix, n);
		printf("\n\n");
		getchar();
		printf("=====================================\n");


	}
	CSC(hipMemcpy(L, L_dev, sizeof(double) * n * n, hipMemcpyDeviceToHost));
	CSC(hipMemcpy(matrix, matrix_dev, sizeof(double) * n * n, hipMemcpyDeviceToHost));
	printf("L-----\n");
	gpu_print_matrix(L, n);
	printf("matrix-----\n");
	gpu_print_matrix(matrix, n);
	
	double det = 1;

	for(int i = 0; i < n; ++i)
	{
		//printf("p[i] = %d ", P[i]);
		printf("L[i] = %lg U[i] = %lg\n", L[i * n + i], matrix[i * n + i]);
		det *= matrix[i * n + i] * L[i * n + i];
	}
	printf("\n");
	printf("det = %f\n", det * sign);
	//printf("matrix-----\n");
	//gpu_print_matrix(matrix, n);


}