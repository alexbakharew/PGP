#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <string.h>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <math.h>
#include <float.h>

#define THREADS_PER_BLOCK 256
#define BLOCKS_PER_GRID 256

#define CSC(call)  					\
do {								\
	hipError_t res = call;			\
	if (res != hipSuccess) {		\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);					\
	}								\
} while(0)

__host__ void gpu_print_matrix(double* matrix, int size)
{
	for (int i = 0; i < size; ++i)
	{
		for (int j = 0; j < size; ++j)
		{
			printf("%.1f ", matrix[i * size + j]);
		}
		printf("\n");
	}
}

__global__ void gpu_transpose(double* matrix, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offsetx = gridDim.x * blockDim.x;
    double temp;
    int curr_row;
    int curr_col;
    while(idx < size * size)
    {
        curr_row = idx / size;
        curr_col = idx % size;
        if(curr_col > curr_row)
        {
            temp = matrix[curr_row * size + curr_col];
            matrix[curr_row * size + curr_col] = matrix[curr_col * size + curr_row];
            matrix[curr_col * size + curr_row] = temp;
        }
	    idx += offsetx;
        
	}
}

__global__ void gpu_swap(double* matrix, int size, int row_from, int row_to)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offsetx = gridDim.x * blockDim.x;
	double tmp;
	for (int i = idx; i < size; i += offsetx)
	{
		tmp = matrix[(i * size) + row_from];
		matrix[(i * size) + row_from] = matrix[(i * size) + row_to];
		matrix[(i * size) + row_to] = tmp;
	}
}

double* multiplication(double* lhs, double* rhs, int n)
{
    double* res = (double*) malloc(sizeof(double) * n * n);

    res = (double*) calloc(n * n, sizeof(double));        


    for(int i = 0; i < n; ++i)
    {
        for(int j = 0; j < n; ++j)
        {        
            for(int t = 0; t < n; ++t)
            {
                res[j * n + i] += lhs[j * n + t] * rhs[t * n + i];
            }
        }
    }
    return res;
}

__global__ void gpu_compute_L(double* matrix, double* L, int size, int curr_row)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offsetx = gridDim.x * blockDim.x;

	for(; idx < size; idx += offsetx)
	{
		if(idx < curr_row)
			continue;

		if(idx == curr_row )
		{
			L[curr_row * size + curr_row] = 1.0;
		}

		else if(fabs(matrix[curr_row * size + curr_row]) > 10e-7)
		{
			L[curr_row * size + idx] = matrix[curr_row * size + idx] / matrix[curr_row * size + curr_row];
		}
	}
}

__global__ void gpu_modify_matrix(double* matrix, double* L, int size, int max_col)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offsetx = gridDim.x * blockDim.x;
	int curr_row;
	int curr_col;
	for(; idx < size * size; idx += offsetx)
	{
		curr_row = idx / size;
		curr_col = idx % size;
		if(curr_col == max_col)
			continue;
		else
		{
			matrix[curr_row * size + curr_col] -= L[max_col * size + curr_col] * matrix[curr_row * size + max_col];
		}
	}
}

struct comparator
{
	__host__ __device__ bool operator()(double lhs, double rhs)
	{
		return fabs(lhs) < fabs(rhs);
	} 
};

int main()
{
	int n;
	scanf("%d", &n);
	double* matrix = (double*)malloc(sizeof(double) * n * n);
	for (int i = 0; i < n * n; ++i)
	{
		scanf("%lf", &matrix[i]);
	}
	double* matrix_dev;
	CSC(hipMalloc(&matrix_dev, sizeof(double) * n * n));
	CSC(hipMemcpy(matrix_dev, matrix, sizeof(double) * n * n, hipMemcpyHostToDevice));

	
	double* L = (double*) calloc(n * n, sizeof(double));
	double* L_dev;
	CSC(hipMalloc(&L_dev, sizeof(double) * n * n));
	CSC(hipMemcpy(L_dev, L, sizeof(double) * n * n, hipMemcpyHostToDevice));
	
	int pos_of_max;
	int sign = 1;
	comparator comp;
	thrust::device_ptr<double> p_matrix = thrust::device_pointer_cast(matrix_dev);
	thrust::device_ptr<double> max_elem;

	hipEvent_t start, end;
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&end));
	CSC(hipEventRecord(start));

	gpu_transpose << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> > (matrix_dev, n);

	for (int row = 0; row < n; ++row)
	{
		max_elem = thrust::max_element(p_matrix + (row * n) + row, p_matrix + ((row + 1) * n), comp);
		pos_of_max = (int)(max_elem - p_matrix) % n;

		if(row != pos_of_max)
		{
			sign *= -1;
			gpu_swap<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>> (matrix_dev, n, row, pos_of_max);
		}

		gpu_compute_L << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> > (matrix_dev, L_dev, n, row);
		gpu_modify_matrix <<< BLOCKS_PER_GRID, THREADS_PER_BLOCK >>> (matrix_dev, L_dev, n, row);
	}
	//
	CSC(hipEventRecord(end));
	CSC(hipEventSynchronize(end));
	float t;
	CSC(hipEventElapsedTime(&t, start, end));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(end));
	printf("GPU time = %.2fms\n", t);

	CSC(hipMemcpy(L, L_dev, sizeof(double) * n * n, hipMemcpyDeviceToHost));
	CSC(hipMemcpy(matrix, matrix_dev, sizeof(double) * n * n, hipMemcpyDeviceToHost));
	
	long double det = 1;
	for(int i = 0; i < n; ++i)
	{
		det *= matrix[i * n + i] * L[i * n + i];
	}
	if(fabs(det) <= 10e-7)
		printf("%.10Lf\n", det);

	else
		printf("%.10Lf\n", det * sign);

	free(matrix);
	free(L);
	hipFree(matrix_dev);	
	hipFree(L_dev);
}