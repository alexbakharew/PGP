#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define NAME_LEN 32

#define CSC(call)  		\
do {								\
	hipError_t res = call;			\
	if (res != hipSuccess) {		\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);					\
	}								\
} while(0)

#define MIN(a,b) a < b ? a : b
#define MAX(a,b) a > b ? a : b

texture<uchar4, 2, hipReadModeElementType> tex;

__global__ void kernel(uchar4 *dst, int w, int h) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int x, y;
	uchar4 p;
	for(x = idx; x < w; x += offsetx) 
	{
		for(y = idy; y < h; y += offsety) 
		{
			p = tex2D(tex, x, y);
			long res = MAX(MAX(p.x, p.y), p.z) + MIN(MIN(p.x, p.y), p.z);
			res /= 2;

			dst[y * w + x].x = res;
			dst[y * w + x].y = res;
			dst[y * w + x].z = res;
			dst[y * w + x].w = p.w;
		}
	}
}

int main(int argc, char* argv[])
{
	if(argc != 3)
	{
		printf("Usage: ./a.out <input data file> <output data file>\n");
		exit(-1);
	}
	char input[NAME_LEN];
	char output[NAME_LEN];
	strcpy(input, argv[1]);
	strcpy( output, argv[2]);

	int width, height;
	FILE *in = fopen(input, "rb");
	if(in == NULL)
	{
		printf("Error while opening input file\n");
		exit(-1);
	}
	fread(&width, sizeof(int), 1 , in);
	fread(&height, sizeof(int), 1 , in);
	uchar4 *image = (uchar4*)malloc(sizeof(uchar4) * width * height);
	fread(image, sizeof(uchar4), width * height, in);
	fclose(in);

	hipArray *arr;
	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
	CSC(hipMallocArray(&arr, &ch, width, height));
	CSC(hipMemcpyToArray(arr, 0, 0, image, sizeof(uchar4) * height * width, hipMemcpyHostToDevice));

	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.channelDesc = ch;
	tex.filterMode = hipFilterModePoint;
	tex.normalized = false; 

	uchar4* new_image = (uchar4*)malloc(sizeof(uchar4) * width * height);

	CSC(hipBindTextureToArray(tex, arr, ch));
	uchar4 *dev_data;
	CSC(hipMalloc(&dev_data, sizeof(uchar4) * height * width));

	hipEvent_t start, end;
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&end));
	CSC(hipEventRecord(start));
	kernel<<<dim3(16, 16), dim3(16, 16)>>>(dev_data, width, height);
	CSC(hipGetLastError());

	CSC(hipEventRecord(end));
	CSC(hipEventSynchronize(end));
	float t;
	CSC(hipEventElapsedTime(&t, start, end));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(end));
	printf("time = %f\n", t);
	
	CSC(hipMemcpy(new_image, dev_data, sizeof(uchar4) * height * width, hipMemcpyDeviceToHost));

	// clock_t start_time = clock();
    // for(int i = 0; i < width; ++i)
    // {
    //     for(int j = 0; j < height; ++j)
    //     {
	// 		int pos = i * height + j;
	// 		long res = MAX(MAX(image[pos].x, image[pos].y), image[pos].z) + MIN(MIN(image[pos].x, image[pos].y), image[pos].z);
	// 		res /= 2;
	// 		new_image[pos].x = res;
	// 		new_image[pos].y = res;
	// 		new_image[pos].z = res;
	// 		new_image[pos].w = image[pos].w;
    //     }
	// }
	// printf("time : %li\n", clock() - start_time);
	
	FILE *out = fopen(argv[2], "wb");
	if(out == NULL)
	{
		printf("Error while opening output file\n");
		free(image);
		free(new_image);
		exit(-1);
	}
	fwrite(&width, sizeof(int), 1, out);
	fwrite(&height, sizeof(int), 1, out);
	fwrite(new_image, sizeof(uchar4), width * height, out);
	fclose(out);
	free(image);
	free(new_image);
}