#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define NAME_LEN 128

#define CSC(call)  		\
do {								\
	hipError_t res = call;			\
	if (res != hipSuccess) {		\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);					\
	}								\
} while(0)

#define MIN(a,b) a < b ? a : b
#define MAX(a,b) a > b ? a : b

texture<uchar4, 2, hipReadModeElementType> tex;

__global__ void kernel(uchar4 *dst, int w, int h) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int x, y;
	uchar4 p;
	for(x = idx; x < w; x += offsetx) 
	{
		for(y = idy; y < h; y += offsety) 
		{
			p = tex2D(tex, x, y);
			unsigned char res = (0.299 * p.x) + (0.587 * p.y) + (0.114 * p.z);

			dst[y * w + x] = make_uchar4(res, res, res, p.w);
		}
	}
}

int main(int argc, char* argv[])
{
	char input[NAME_LEN];
	char output[NAME_LEN];

	scanf("%s", input);
	scanf("%s", output);

	int width, height;
	FILE *in = fopen(input, "rb");
	if(in == NULL)
	{
		printf("Error while opening input file\n");
		exit(-1);
	}
	fread(&width, sizeof(int), 1 , in);
	fread(&height, sizeof(int), 1 , in);
	uchar4 *image = (uchar4*)malloc(sizeof(uchar4) * width * height);
	fread(image, sizeof(uchar4), width * height, in);
	fclose(in);

	hipArray *arr;
	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
	CSC(hipMallocArray(&arr, &ch, width, height));
	CSC(hipMemcpyToArray(arr, 0, 0, image, sizeof(uchar4) * height * width, hipMemcpyHostToDevice));

	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.channelDesc = ch;
	tex.filterMode = hipFilterModePoint;
	tex.normalized = false; 

	uchar4* new_image = (uchar4*)malloc(sizeof(uchar4) * width * height);

	CSC(hipBindTextureToArray(tex, arr, ch));
	uchar4 *dev_data;
	CSC(hipMalloc(&dev_data, sizeof(uchar4) * height * width));

	hipEvent_t start, end;
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&end));
	CSC(hipEventRecord(start));
	kernel<<<dim3(32, 32), dim3(32, 32)>>>(dev_data, width, height);
	CSC(hipGetLastError());

	CSC(hipEventRecord(end));
	CSC(hipEventSynchronize(end));
	float t;
	CSC(hipEventElapsedTime(&t, start, end));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(end));
	printf("GPU time = %.2fms\n", t);

	CSC(hipMemcpy(new_image, dev_data, sizeof(uchar4) * height * width, hipMemcpyDeviceToHost));

	clock_t start_time = clock();
    for(int i = 0; i < height; ++i)
    {
        for(int j = 0; j < width; ++j)
        {
			int pos = i * width + j;
			long res = MAX(MAX(image[pos].x, image[pos].y), image[pos].z) + MIN(MIN(image[pos].x, image[pos].y), image[pos].z);
			res /= 2;
			new_image[pos].x = res;
			new_image[pos].y = res;
			new_image[pos].z = res;
			new_image[pos].w = image[pos].w;
        }
	}
	printf("CPU time = %.2fms\n", (double)(clock() - start_time) * 1000 /CLOCKS_PER_SEC);

	FILE *out = fopen(output, "wb");
	if(out == NULL)
	{
		printf("Error while opening output file\n");
		free(image);
		free(new_image);
		exit(-1);
	}
	fwrite(&width, sizeof(int), 1, out);
	fwrite(&height, sizeof(int), 1, out);
	fwrite(new_image, sizeof(uchar4), width * height, out);

	hipUnbindTexture(tex);
	hipFreeArray(arr);
	hipFree(dev_data);
	fclose(out);
	free(image);
	free(new_image);
}