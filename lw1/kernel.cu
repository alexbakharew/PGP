#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

using ll = long long;
#define CSC(call)  					\
do {								\
	hipError_t res = call;			\
	if (res != hipSuccess) {		\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);					\
	}								\
} while(0)



void RunOnCPU(const double* arr1, const double* arr2, const llu size)
{
	int* result = (int*) malloc(size * sizeof(int));
	for (llu i = 0; i < size; ++i)
	{
		result[i] = (arr1[i] > arr2[i] ? arr1[i] : arr2[i]);
	}
}

__global__ void kernel(const double* arr1, const double* arr2, double* result, const ll size)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	while (idx < size)
	{
		result[idx] = (arr1[idx] > arr2[idx] ? arr1[idx] : arr2[idx]);
		idx += offset;
	}
}


void RunOnGPU(const double* arr1, const double* arr2, const llu n)
{


}

int main() 
{
	ll n;
	double* arr1;
	double* arr2;
	scanf("%lli", &n);
	if (n < 0)
	{
		printf("ERROR: negative size of vectors. Exit\n");
		return 0;
	}
	arr1 = (double*)(malloc(n * sizeof(double)));
	arr2 = (double*)(malloc(n * sizeof(double)));
	for (int i = 0; i < n; ++i)
	{
		scanf("%lf", &arr1[i]);
	}

	for (int i = 0; i < n; ++i)
	{
		scanf("%lf", &arr2[i]);
	}
	//calc time
	RunOnCPU(arr1, arr2, n);
	
	double* dev_arr1;
	double* dev_arr2;
	double* result_on_gpu;

	CSC(hipMalloc(&dev_arr1, sizeof(double) * n));
	CSC(hipMalloc(&dev_arr2, sizeof(double) * n));
	CSC(hipMalloc(&result_on_gpu, sizeof(double) * n));

	CSC(hipMemcpy(dev_arr1, arr1, sizeof(double) * n, hipMemcpyHostToDevice));
	CSC(hipMemcpy(dev_arr2, arr2, sizeof(double) * n, hipMemcpyHostToDevice));

	//calc time
	kernel << <256, 256 >> > (dev_arr1, dev_arr2, result_on_gpu, n);

	double* result_on_cpu = (double*)malloc(sizeof(double) * n);

	CSC(hipMemcpy(result_on_cpu, result_on_gpu, sizeof(double) * n, hipMemcpyDeviceToHost));
	CSC(hipGetLastError());

	for (ll i = 0; i < n; ++i)
	{
		std::cout << std::fixed << std::scientific;

		std::cout << std::setprecision(10) << result_on_cpu[i];
		if (i < n - 1)
			std::cout << " ";
		else
			std::cout << std::endl;
	}

	free(arr1);
	free(arr2);
	free(result_on_cpu);
	CSC(hipFree(dev_arr1));
	CSC(hipFree(dev_arr2));
	CSC(hipFree(result_on_gpu));

	return 0;
}
